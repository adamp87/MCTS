#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <mutex>
#include <iostream>

#include "defs.hpp"
#include "mcts.cuh"
#include "hearts.hpp"

// NOTE: this is a dirty solution, compile flow.cpp for cuda here
#include "flownetwork.cpp"

struct RolloutCUDA::impl {
    Hearts* u_state;
    Hearts::Player* u_player;
    uint8* u_result; //players(4)*maxrollout
    hiprandState* d_rnd;

    std::mutex lock;
    constexpr static int nThread = 32;
};

__global__ void rollout(const Hearts* u_state,
                        const Hearts::Player* u_player,
                        hiprandState* d_rnd,
                        uint8* u_result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint8 cards[52];
    Hearts state(*u_state);
    Hearts::Player player(*u_player);

    // Copy state to local memory for efficiency
    hiprandState localRnd = d_rnd[idx];

    while (!state.isGameOver()) {
        uint8 count = state.getPossibleCards(player, cards);
        uint8 pick = hiprand(&localRnd) % count;
        state.update(cards[pick]);
    }

    // Copy state back to global memory
    d_rnd[idx] = localRnd;

    state.computePoints(u_result + 4 * idx);
}

__global__ void setup_random(hiprandState* state, unsigned int seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(seed, id, 0, &state[id]);
}

RolloutCUDA::RolloutCUDA(unsigned int* iterations, unsigned int seed) {
    pimpl = 0;
    int deviceCount = 0;
    if(hipGetDeviceCount(&deviceCount) != hipSuccess) {
        std::cout << "Failed to get device count" << std::endl;
        return;
    }
    if (deviceCount == 0) {
        std::cout << "No CUDA device found" << std::endl;
        return;
    }

    unsigned int maxIterations = iterations[0];
    for (int i = 0; i < 4; ++i) {
        if (iterations[i] == 1)
            continue; // cuda rollout disabled for player
        if (iterations[i] % RolloutCUDA::impl::nThread != 0) {
            std::cout << "Rollout " << i << " is not dividable with " << RolloutCUDA::impl::nThread << std::endl;
            return;
        }
        if (maxIterations < iterations[i]) {
            maxIterations = iterations[i];
        }
    }

    std::unique_ptr<RolloutCUDA::impl> ptr(new RolloutCUDA::impl());
    dim3 threads(RolloutCUDA::impl::nThread);
    dim3 blocks(maxIterations / threads.x);
    if(hipMallocManaged(&ptr->u_state, sizeof(Hearts)) != hipSuccess) {
        std::cout << "Failed to allocate state" << std::endl;
        return;
    }
    if(hipMallocManaged(&ptr->u_player, sizeof(Hearts::Player)) != hipSuccess) {
        std::cout << "Failed to allocate player" << std::endl;
        return;
    }
    if(hipMalloc(&ptr->d_rnd, sizeof(hiprandState) * maxIterations) != hipSuccess) {
        std::cout << "Failed to allocate random" << std::endl;
        return;
    }
    if(hipMallocManaged(&ptr->u_result, sizeof(uint8) * 4 * maxIterations) != hipSuccess) {
        std::cout << "Failed to allocate results" << std::endl;
        return;
    }

    setup_random<<<blocks, threads>>>(ptr->d_rnd, seed);
    if (hipDeviceSynchronize() != hipSuccess) {
        std::cout << "Failed to sync CUDA call" << std::endl;
        return;
    }

    pimpl = ptr.release();
}

RolloutCUDA::~RolloutCUDA() {
    if (pimpl == 0)
        return;
    hipFree(pimpl->d_rnd);
    hipFree(pimpl->u_state);
    hipFree(pimpl->u_player);
    hipFree(pimpl->u_result);
    delete pimpl;
    pimpl = 0;
}

__host__ bool RolloutCUDA::cuRollout(const Hearts& state,
                                     const Hearts::Player& player,
                                     unsigned int iterations,
                                     std::vector<uint8>& points) const {
    std::unique_lock<std::mutex> lock(pimpl->lock, std::defer_lock);
    if (lock.try_lock() == false)
        return false;
    dim3 threads(RolloutCUDA::impl::nThread);
    dim3 blocks(iterations / threads.x);
    *pimpl->u_state = state;
    *pimpl->u_player = player;
    rollout<<<blocks, threads>>>(pimpl->u_state,
                                 pimpl->u_player,
                                 pimpl->d_rnd,
                                 pimpl->u_result);
    if(hipDeviceSynchronize() != hipSuccess) {
        std::cout << "Failed to sync CUDA call" << std::endl;
        return false;
    }

    points.clear();
    points.insert(points.end(), pimpl->u_result, pimpl->u_result + 4 * iterations);
    return true;
}

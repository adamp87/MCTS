#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <mutex>
#include <numeric>
#include <iostream>

#include "mcts.cuh"
#include "hearts.hpp"

// NOTE: this is a dirty solution, compile flow.cpp for cuda here
#include "flownetwork.cpp"

class Hearts;
template class RolloutCUDA<Hearts>;

template <class TProblem>
struct RolloutCUDA<TProblem>::impl {
    TProblem* u_state;
    double* u_result; //size=maxrollout
    hiprandState* d_rnd;

    std::mutex lock;
    constexpr static int nThread = 32;
};

template <typename TProblem>
__global__ void rollout(int idxAi,
                        int maxRolloutDepth,
                        const TProblem* u_state,
                        hiprandState* d_rnd,
                        double* u_result) {
    typedef typename TProblem::ActType ActType;
    typedef typename TProblem::ActCounterType ActCounterType;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    TProblem state(*u_state);
    ActType actions[TProblem::MaxActions];

    // Copy state to local memory for efficiency
    hiprandState localRnd = d_rnd[idx];

    int depth = 0; //if max is zero, until finished
    while (!state.isFinished()) {
        if (++depth == maxRolloutDepth)
            break;
        ActCounterType count = state.getPossibleActions(idxAi, state.getPlayer(), actions);
        ActCounterType pick = hiprand(&localRnd) % count;
        state.update(actions[pick]);
    }

    // Copy state back to global memory
    d_rnd[idx] = localRnd;

    u_result[idx] = state.computeMCTSWin(idxAi);
}

__global__ void setup_random(hiprandState* state, unsigned int seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(seed, id, 0, &state[id]);
}

template <class TProblem>
RolloutCUDA<TProblem>::RolloutCUDA(unsigned int* iterations, unsigned int seed) {
    pimpl = 0;
    int deviceCount = 0;
    if(hipGetDeviceCount(&deviceCount) != hipSuccess) {
        std::cout << "Failed to get device count" << std::endl;
        return;
    }
    if (deviceCount == 0) {
        std::cout << "No CUDA device found" << std::endl;
        return;
    }
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (i == 0)
            std::cout << "Selected ";
        std::cout << "Device number: " << i << " Device name: " << prop.name << std::endl;
    }

    unsigned int maxIterations = iterations[0];
    for (int i = 0; i < 4; ++i) {
        if (iterations[i] == 1)
            continue; // cuda rollout disabled for player
        if (iterations[i] % RolloutCUDA::impl::nThread != 0) {
            std::cout << "Rollout " << i << " is not dividable with " << RolloutCUDA::impl::nThread << std::endl;
            return;
        }
        if (maxIterations < iterations[i]) {
            maxIterations = iterations[i];
        }
    }
    if (maxIterations == 1) {
        std::cout << "CUDA was not requested, enable with e.g. (r3 2048)" << std::endl;
        return;
    }

    std::unique_ptr<RolloutCUDA::impl> ptr(new RolloutCUDA::impl());
    dim3 threads(RolloutCUDA::impl::nThread);
    dim3 blocks(maxIterations / threads.x);
    if(hipMallocManaged(&ptr->u_state, sizeof(TProblem)) != hipSuccess) {
        std::cout << "Failed to allocate state" << std::endl;
        return;
    }
    if(hipMalloc(&ptr->d_rnd, sizeof(hiprandState) * maxIterations) != hipSuccess) {
        std::cout << "Failed to allocate random" << std::endl;
        return;
    }
    if(hipMallocManaged(&ptr->u_result, sizeof(double) * maxIterations) != hipSuccess) {
        std::cout << "Failed to allocate results" << std::endl;
        return;
    }

    setup_random<<<blocks, threads>>>(ptr->d_rnd, seed);
    if (hipDeviceSynchronize() != hipSuccess) {
        std::cout << "Failed to sync CUDA call" << std::endl;
        return;
    }

    pimpl = ptr.release();
}

template <class TProblem>
RolloutCUDA<TProblem>::~RolloutCUDA() {
    if (pimpl == 0)
        return;
    hipFree(pimpl->d_rnd);
    hipFree(pimpl->u_state);
    hipFree(pimpl->u_result);
    delete pimpl;
    pimpl = 0;
}

template <typename TProblem>
__host__ bool RolloutCUDA<TProblem>::cuRollout(int idxAi,
                                               int maxRolloutDepth,
                                               const TProblem& state,
                                               unsigned int iterations,
                                               double& winSum) const {
    std::unique_lock<std::mutex> lock(pimpl->lock, std::defer_lock);
    if (lock.try_lock() == false)
        return false;
    dim3 threads(RolloutCUDA::impl::nThread);
    dim3 blocks(iterations / threads.x);
    *pimpl->u_state = state;
    rollout<<<blocks, threads>>>(idxAi,
                                 maxRolloutDepth,
                                 pimpl->u_state,
                                 pimpl->d_rnd,
                                 pimpl->u_result);
    if(hipDeviceSynchronize() != hipSuccess) {
        std::cout << "Failed to sync CUDA call" << std::endl;
        return false;
    }

    winSum = std::accumulate(pimpl->u_result, pimpl->u_result + iterations, 0.0);
    return true;
}

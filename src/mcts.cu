#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "defs.hpp"
#include "mcts.cuh"
#include "hearts.hpp"

__host__ unsigned int* cuRollout(const Hearts::State& state,
                                 const Hearts::Player& ai,
                                 const uint8* cards,
                                 uint8 nCards,
                                 RolloutContainer* data) {
    return 0; // implement
}

__host__ RolloutContainer* init(uint32 rollout) {
    return 0; // implement
}

__host__ void free(RolloutContainer* data) {
    // implement
}

RolloutContainerCPP::RolloutContainerCPP(uint32 iterations) {
    data = init(iterations);
}

RolloutContainerCPP::~RolloutContainerCPP() {
    free(data);
    data = 0;
}

unsigned int* RolloutContainerCPP::_cuRollout(const Hearts::State& state,
                                              const Hearts::Player& ai,
                                              const uint8* cards,
                                              uint8 nCards) {
    return cuRollout(state, ai, cards, nCards, data);
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "defs.hpp"
#include "mcts.cuh"
#include "hearts.hpp"

struct RolloutContainer {
    struct Input {
        Hearts::State state;
        Hearts::Player ai;
        uint8 cards[52];
        uint8 nCards;
    };

    Input* u_input;
    hiprandState* d_rnd;
    unsigned int* u_result; //52*28- cards*wincount, atomic

    unsigned int threadCount;
    unsigned int blockCountPerCard;
};

__global__ void rollout(const RolloutContainer::Input* src,
                        hiprandState* rnd,
                        unsigned int* dst) {
    uint cardIdx = blockIdx.x / (gridDim.x / src->nCards);
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint8 cards[52];
    Hearts::Player ai(src->ai);
    Hearts::State state(src->state);
    Hearts::update(state, src->cards[cardIdx]);

    // Copy state to local memory for efficiency
    hiprandState localRnd = rnd[idx];

    while (!state.isTerminal()) {
        uint8 count = Hearts::getPossibleCards(state, ai, cards);
        uint8 pick = hiprand(&localRnd) % count;
        Hearts::update(state, cards[pick]);
    }

    // Copy state back to global memory
    rnd[idx] = localRnd;

    uint8 points[4];
    Hearts::computePoints(state, points);
    uint* address = dst + cardIdx * 28 + Hearts::mapPoints2Wins(ai, points);
    atomicAdd(address, 1);
}

__global__ void setup_random(hiprandState* state, unsigned int seed) {
    uint id = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(seed, id, 0, &state[id]);
}

__host__ unsigned int* cuRollout(const Hearts::State& state,
                                 const Hearts::Player& ai,
                                 const uint8* cards,
                                 uint8 nCards,
                                 RolloutContainer* data) {
    data->u_input->state = state;
    data->u_input->ai = ai;
    std::copy(cards, cards + nCards, data->u_input->cards);
    data->u_input->nCards = nCards;
    std::fill(data->u_result, data->u_result + 52 * 28, 0);

    dim3 blocks(data->blockCountPerCard * nCards);
    dim3 threads(data->threadCount);
    rollout<<<blocks, threads>>>(data->u_input, data->d_rnd, data->u_result);
    hipDeviceSynchronize();

    return data->u_result;
}

__host__ RolloutContainer* initData(uint32 rollout, unsigned int seed) {
    int deviceCount = 0;
    if(hipGetDeviceCount(&deviceCount) != hipSuccess) {
        std::cout << "Failed to get device count" << std::endl;
        return 0;
    }
    if (deviceCount == 0) {
        std::cout << "No CUDA device found" << std::endl;
        return 0;
    }

    if ((rollout & (rollout - 1)) != 0) {
        std::cout << "Not power of two" << std::endl;
        return 0;
    }

    std::unique_ptr<RolloutContainer> data(new RolloutContainer());
    data->threadCount = 32;
    data->blockCountPerCard = rollout / data->threadCount;
    uint maxThreads = data->blockCountPerCard * data->threadCount * 52;
    if(hipMallocManaged(&data->u_input, sizeof(RolloutContainer::Input)) != hipSuccess) {
        std::cout << "Failed to allocate input" << std::endl;
        return 0;
    }
    if(hipMalloc(&data->d_rnd, sizeof(hiprandState) * maxThreads) != hipSuccess) {
        std::cout << "Failed to allocate random" << std::endl;
        return 0;
    }
    if(hipMallocManaged(&data->u_result, sizeof(unsigned int) * 52 * 28) != hipSuccess) {
        std::cout << "Failed to allocate results" << std::endl;
        return 0;
    }

    setup_random<<<data->blockCountPerCard * 52, data->threadCount>>>(data->d_rnd, seed);
    hipDeviceSynchronize();

    return data.release();
}

__host__ void freeData(RolloutContainer* data) {
    if (data == 0)
        return;
    hipFree(data->d_rnd);
    hipFree(data->u_input);
    hipFree(data->u_result);
}

RolloutContainerCPP::RolloutContainerCPP(uint32 iterations, unsigned int seed) {
    data = initData(iterations, seed);
}

RolloutContainerCPP::~RolloutContainerCPP() {
    freeData(data);
    data = 0;
}

unsigned int* RolloutContainerCPP::_cuRollout(const Hearts::State& state,
                                              const Hearts::Player& ai,
                                              const uint8* cards,
                                              uint8 nCards) {
    return cuRollout(state, ai, cards, nCards, data);
}
